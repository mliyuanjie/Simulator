#include "hip/hip_runtime.h"
﻿#include "solvercuda.cuh"
#include <hipsolver.h>
#include <chrono>
#include <iomanip>
#include <fstream>

__global__ void checkSymmetry(const hipDoubleComplex* A, int n, bool* isSymmetric) {
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n && j < n && i < j) {
		double x1 = A[i * n + j].x - A[j * n + i].x;
		double x2 = A[i * n + j].y - A[j * n + i].y;
		if ((abs(x1) + abs(x2)) > 1e-8) {
			*isSymmetric = false;
		}
	}
}

void Solver::Schur() {
	size_t allsize = sizeA + sizeV * 2 + sizeS + lu_work_size * sizeof(hipDoubleComplex);
	allsize /= 1024 * 1024;
	std::cout << "|---Schur---|---GPU---" << std::fixed << std::setprecision(2) << 
		double(allsize) / 1024.0 << " GB---|" << std::endl;
	hipDoubleComplex alpha = make_hipDoubleComplex(-1.0, 0.0);
	hipDoubleComplex beta = make_hipDoubleComplex(1.0, 0.0);
	hipMemset(d_A, 0, sizeA);
	hipMemset(d_B, 0, sizeS);
	hipMemset(d_y, 0, sizeV);
	hipMemset(d_x, 0, sizeV);

	matrix.compute_matrix(d_A, 0, NpA, 0, NpA, false);
	matrix.compute_matrix(d_B, 0, NpA, NpA, Np, false);
	matrix.right_side_F(d_y, 0, NpA);
	matrix.right_side_F(d_y + offsetS, NpA, Np);

	// [A  B]  [x1]  ->  [b1]
	// [C  D]  [x2]  ->  [b2]
	//LU->A
	hipsolverDnZgetrf(cusolverH, dimA, dimA, d_A, dimA, lu_work, devIpiv, devInfo);
	//b1 = A'b1
	hipsolverDnZgetrs(cusolverH, HIPBLAS_OP_N, dimA, cols, d_A, dimA, devIpiv, d_y, dimA, devInfo);
	//B = A'B
	hipsolverDnZgetrs(cusolverH, HIPBLAS_OP_N, dimA, dimS, d_A, dimA, devIpiv, d_B, dimA, devInfo);
	//D -> A
	matrix.compute_matrix(d_A, NpA, Np, NpA, Np, false);
	matrix.compute_matrix(lu_work, NpA, Np, 0, NpA, false);
	//A = A - C * B
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimS, dimS, dimA,
		&alpha, lu_work, dimS, d_B, dimA, &beta, d_A, dimS);
	//b2 = b2 - C * b1
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimS, cols, dimA,
		&alpha, lu_work, dimS, d_y, dimA, &beta, d_y + offsetS, dimS);
	//LU->S
	hipsolverDnZgetrf(cusolverH, dimS, dimS, d_A, dimS, lu_work, devIpiv, devInfo);
	//solve x2
	hipsolverDnZgetrs(cusolverH, HIPBLAS_OP_N, dimS, cols, d_A, dimS, devIpiv, d_y + offsetS, dimS, devInfo);
	//x1 = b1 - B * x2
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimA, cols, dimS,
		&alpha, d_B, dimA, d_y + offsetS, dimS, &beta, d_y, dimA);
	hipMemcpy2D(d_x, dim * sizeof(hipDoubleComplex),
		d_y, dimA * sizeof(hipDoubleComplex), dimA * sizeof(hipDoubleComplex), cols,
		hipMemcpyDeviceToDevice);
	hipMemcpy2D(d_x + dimA, dim * sizeof(hipDoubleComplex),
		d_y + offsetS, dimS * sizeof(hipDoubleComplex), dimS * sizeof(hipDoubleComplex), cols,
		hipMemcpyDeviceToDevice);
	matrix.right_side_F(d_y, 0, Np);
}

void Solver::LU() {
	std::cout << "|---LU---|---GPU---" << 
		std::fixed << std::setprecision(2) << 
		double(sizeA + sizeV * 2) / 1024.0 / 1024.0 / 1024.0 + 16.0 * 
		double(lu_work_size) / 1024.0 / 1024.0 / 1024.0 << " GB---|" << std::endl;
	matrix.compute_matrix(d_A, 0, Np, 0, Np, false);
	
	matrix.right_side_F(d_x, 0, Np);
	
	hipsolverDnZgetrf(cusolverH, dim, dim, d_A, dim, lu_work, devIpiv, devInfo);

	int h_devInfo = 0;
	hipMemcpy(&h_devInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	if (h_devInfo != 0) {
		std::cerr << "|---Solving linear system failed " << h_devInfo << "---|" << std::endl;
	}
	hipsolverDnZgetrs(cusolverH, HIPBLAS_OP_N, dim, cols, d_A, dim, devIpiv, d_x, dim, devInfo);
	h_devInfo = 0;
	hipMemcpy(&h_devInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	if (h_devInfo != 0) {
		std::cerr << "|---Solving linear system failed " << h_devInfo << "---|" << std::endl;
	}
	matrix.right_side_F(d_y, 0, Np);
}

void Solver::LU_mkl() {
	std::cout << "|---LU---|---CPU---" << std::fixed << std::setprecision(2) << 
		double(dim * dim * 16 + sizeV) / 1024.0 / 1024.0 / 1024.0 << " GB---|" << std::endl;
	size_t dpitch = dim * sizeof(hipDoubleComplex);
	for (int i = 0; i < Np; i += 100) {
		int si = i;
		int ei = (i + 100 <= Np) ? i + 100 : Np;
		size_t spitch = (ei - si) * nharm * sizeof(hipDoubleComplex);
		for (int j = 0; j < Np; j += 100) {
			int sj = j;
			int ej = (j + 100 <= Np) ? j + 100 : Np;
			matrix.compute_matrix(d_A, si, ei, sj, ej, false);
			size_t offset = dim * j * nharm + i * nharm;
			hipError_t err = hipMemcpy2D((void*)(c_A + offset), dpitch, d_A, spitch, spitch, (ej - sj) * nharm, hipMemcpyDeviceToHost);
		}
	}
	matrix.right_side_F_mkl(c_x, 0, Np);
	std::vector<int> ipiv(dim);
	LAPACKE_zgetrf(LAPACK_COL_MAJOR, dim, dim, c_A, dim, ipiv.data());
	LAPACKE_zgetrs(LAPACK_COL_MAJOR, 'N', dim, cols, c_A, dim, ipiv.data(), c_x, dim);
	hipMemcpy(d_x, c_x, sizeV, hipMemcpyHostToDevice);
	matrix.right_side_F(d_y, 0, Np);
}

void Solver::verify(std::string& fn){
	hipDoubleComplex alpha = make_hipDoubleComplex(-1.0, 0.0);
	hipDoubleComplex beta = make_hipDoubleComplex(1.0, 0.0);
	matrix.right_side_F(d_y, 0, Np);
	matrix.matmult(d_x, -1, d_y, 1);
	double normmc = 1.0;
	hipblasDznrm2(handle, dim * cols, d_y, 1, &normmc);
	std::cout << "| verify | norm2: " << normmc << " |" << std::endl;
	if (fn == "")
		return;
	hipDoubleComplex* M_t;
	std::vector<std::complex<double>> M(dim * dim);
	hipMalloc(&M_t, dim * dim * sizeof(hipDoubleComplex));
	matrix.compute_matrix(M_t, 0, Np, 0, Np, false);
	hipMemcpy(M.data(), M_t, dim * dim * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	//compare_matrix_F(fn, dim, M, 1e-8);
	hipFree(M_t);
}

void Solver::solve(const char* str) {
	auto start = std::chrono::high_resolution_clock::now();
	if (solver_type == 0)
		LU();
	else if (solver_type == 2)
		Schur();
	else if (solver_type == 1)
		LU_mkl();
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
	std::cout << "|---" << str <<"---|---" << duration.count() / 1000.0 << " s---|" << std::endl;
	//hipMemcpy(x_c.data(), d_x, dim * cols * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
}

void Solver::matmult(double alpha, double beta) {
	matrix.matmult(d_x, alpha, d_y, beta);
}

void Solver::clear() {
	if (d_A) {
		hipFree(d_A); d_A = nullptr;
	}
	if (d_B) {
		hipFree(d_B); d_B = nullptr;
	}
	if (d_x) {
		hipFree(d_x); d_x = nullptr;
	}
	if (d_y) {
		hipFree(d_y); d_y = nullptr;
	}
	if (lu_work) {
		hipFree(lu_work); lu_work = nullptr;
	}
	if (devIpiv) {
		hipFree(devIpiv); devIpiv = nullptr;
	}
	if (c_A) {
		delete[] c_A;
		c_A = nullptr;
	}
	if (c_x) {
		delete[] c_x;
		c_x = nullptr;
	}
}

void Solver::initial() {
	if (solver_type == 2) {
		hipMalloc(&d_A, sizeA);
		hipMalloc(&d_B, sizeS);
		hipMalloc(&d_y, sizeV);
		hipMalloc(&d_x, sizeV);
		hipMalloc(&lu_work, lu_work_size * sizeof(hipDoubleComplex));
		hipMalloc(&devIpiv, dimA * sizeof(int));
	}
	else if (solver_type == 0) {
		hipMalloc(&d_A, sizeA);
		hipMalloc(&d_y, sizeV);
		hipMalloc(&d_x, sizeV);
		hipsolverDnZgetrf_bufferSize(cusolverH, dim, dim, d_A, dim, &lu_work_size);
		hipMalloc(&lu_work, lu_work_size * sizeof(hipDoubleComplex));
		hipMalloc(&devIpiv, dim * sizeof(int));
	}
	else if (solver_type == 1) {
		hipMalloc(&d_A, sizeA);
		hipMalloc(&d_y, sizeV);
		hipMalloc(&d_x, sizeV);
		c_A = new MKL_Complex16[dim * dim];
		c_x = new MKL_Complex16[dim * cols];
	}
}
